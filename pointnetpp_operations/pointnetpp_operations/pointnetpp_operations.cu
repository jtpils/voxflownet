#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>

#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>

#define CUDA_CHECK_ERRORS()                                           \
  do {                                                                \
    hipError_t err = hipGetLastError();                             \
    if (hipSuccess != err) {                                         \
      fprintf(stderr, "CUDA kernel failed : %s\n%s at L:%d in %s\n",  \
              hipGetErrorString(err), __PRETTY_FUNCTION__, __LINE__, \
              __FILE__);                                              \
      exit(-1);                                                       \
    }                                                                 \
  } while (0)


#define TOTAL_THREADS 512

inline int opt_n_threads(const int work_size) {
    const int pow_2 = std::log(static_cast<double>(work_size)) / std::log(2.0);

    return max(min(1 << pow_2, TOTAL_THREADS), 1);
}

inline dim3 opt_block_config(const int x, const int y) {
    const int x_threads = opt_n_threads(x);
    const int y_threads = max(min(opt_n_threads(y), TOTAL_THREADS / x_threads), 1);
    dim3 block_config(x_threads, y_threads, 1);

    return block_config;
}



// input: points(b, c, n) idx(b, m)
// output: out(b, c, m)
__global__ void gather_points_kernel(int b, int c, int n, int m,
                                     const float *__restrict__ points,
                                     const int *__restrict__ idx,
                                     float *__restrict__ out) {
  for (int i = blockIdx.x; i < b; i += gridDim.x) {
    for (int l = blockIdx.y; l < c; l += gridDim.y) {
      for (int j = threadIdx.x; j < m; j += blockDim.x) {
        int a = idx[i * m + j];
        out[(i * c + l) * m + j] = points[(i * c + l) * n + a];
      }
    }
  }
}

void GatherPointsKernelLauncher(const int b, const int c, const int n, const int npoints,
                                const float *points, const int *idx, float *out) {
  gather_points_kernel<<<dim3(b, c, 1), opt_n_threads(npoints), 0,
                         at::cuda::getCurrentCUDAStream()>>>(b, c, n, npoints, points, idx, out);
  CUDA_CHECK_ERRORS();
}
// input: grad_out(b, c, m) idx(b, m)
// output: grad_points(b, c, n)
__global__ void gather_points_grad_kernel(int b, int c, int n, int m,
                                          const float *__restrict__ grad_out,
                                          const int *__restrict__ idx,
                                          float *__restrict__ grad_points) {
  for (int i = blockIdx.x; i < b; i += gridDim.x) {
    for (int l = blockIdx.y; l < c; l += gridDim.y) {
      for (int j = threadIdx.x; j < m; j += blockDim.x) {
        int a = idx[i * m + j];
        atomicAdd(grad_points + (i * c + l) * n + a,
                  grad_out[(i * c + l) * m + j]);
      }
    }
  }
}
void GatherPointsGradKernelLauncher(const int b, const int c, const int n, const int npoints,
                                    const float *grad_out, const int *idx, float *grad_points) {
  gather_points_grad_kernel<<<dim3(b, c, 1), opt_n_threads(npoints), 0,
                              at::cuda::getCurrentCUDAStream()>>>(
      b, c, n, npoints, grad_out, idx, grad_points);
  CUDA_CHECK_ERRORS();
}
__device__ void __update(float *__restrict__ dists, int *__restrict__ dists_i,
                         int idx1, int idx2) {
  const float v1 = dists[idx1], v2 = dists[idx2];
  const int i1 = dists_i[idx1], i2 = dists_i[idx2];
  dists[idx1] = max(v1, v2);
  dists_i[idx1] = v2 > v1 ? i2 : i1;
}
// Input dataset: (b, n, 3), tmp: (b, n)
// Ouput idxs (b, m)
template <unsigned int block_size>
__global__ void furthest_point_sampling_kernel(
    int b, int n, int m, const float *__restrict__ dataset,
    float *__restrict__ temp, int *__restrict__ idxs) {
  if (m <= 0) return;
  __shared__ float dists[block_size];
  __shared__ int dists_i[block_size];
  int batch_index = blockIdx.x;
  dataset += batch_index * n * 3;
  temp += batch_index * n;
  idxs += batch_index * m;
  int tid = threadIdx.x;
  const int stride = block_size;
  int old = 0;
  if (threadIdx.x == 0) idxs[0] = old;
  __syncthreads();
  for (int j = 1; j < m; j++) {
    int besti = 0;
    float best = -1;
    float x1 = dataset[old * 3 + 0];
    float y1 = dataset[old * 3 + 1];
    float z1 = dataset[old * 3 + 2];
    for (int k = tid; k < n; k += stride) {
      float x2, y2, z2;
      x2 = dataset[k * 3 + 0];
      y2 = dataset[k * 3 + 1];
      z2 = dataset[k * 3 + 2];
      float mag = (x2 * x2) + (y2 * y2) + (z2 * z2);
      if (mag <= 1e-3) continue;
      float d =
          (x2 - x1) * (x2 - x1) + (y2 - y1) * (y2 - y1) + (z2 - z1) * (z2 - z1);
      float d2 = min(d, temp[k]);
      temp[k] = d2;
      besti = d2 > best ? k : besti;
      best = d2 > best ? d2 : best;
    }
    dists[tid] = best;
    dists_i[tid] = besti;
    __syncthreads();
    if (block_size >= 512) {
      if (tid < 256) {
        __update(dists, dists_i, tid, tid + 256);
      }
      __syncthreads();
    }
    if (block_size >= 256) {
      if (tid < 128) {
        __update(dists, dists_i, tid, tid + 128);
      }
      __syncthreads();
    }
    if (block_size >= 128) {
      if (tid < 64) {
        __update(dists, dists_i, tid, tid + 64);
      }
      __syncthreads();
    }
    if (block_size >= 64) {
      if (tid < 32) {
        __update(dists, dists_i, tid, tid + 32);
      }
      __syncthreads();
    }
    if (block_size >= 32) {
      if (tid < 16) {
        __update(dists, dists_i, tid, tid + 16);
      }
      __syncthreads();
    }
    if (block_size >= 16) {
      if (tid < 8) {
        __update(dists, dists_i, tid, tid + 8);
      }
      __syncthreads();
    }
    if (block_size >= 8) {
      if (tid < 4) {
        __update(dists, dists_i, tid, tid + 4);
      }
      __syncthreads();
    }
    if (block_size >= 4) {
      if (tid < 2) {
        __update(dists, dists_i, tid, tid + 2);
      }
      __syncthreads();
    }
    if (block_size >= 2) {
      if (tid < 1) {
        __update(dists, dists_i, tid, tid + 1);
      }
      __syncthreads();
    }
    old = dists_i[0];
    if (tid == 0) idxs[j] = old;
  }
}
void FurthestPointSamplingKernelLauncher(const int b, const int n, const int m, const float *dataset, float *temp, int *idxs) {
  unsigned int n_threads = opt_n_threads(n);
  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  switch (n_threads) {
    case 512:
      furthest_point_sampling_kernel<512>
          <<<b, n_threads, 0, stream>>>(b, n, m, dataset, temp, idxs);
      break;
    case 256:
      furthest_point_sampling_kernel<256>
          <<<b, n_threads, 0, stream>>>(b, n, m, dataset, temp, idxs);
      break;
    case 128:
      furthest_point_sampling_kernel<128>
          <<<b, n_threads, 0, stream>>>(b, n, m, dataset, temp, idxs);
      break;
    case 64:
      furthest_point_sampling_kernel<64>
          <<<b, n_threads, 0, stream>>>(b, n, m, dataset, temp, idxs);
      break;
    case 32:
      furthest_point_sampling_kernel<32>
          <<<b, n_threads, 0, stream>>>(b, n, m, dataset, temp, idxs);
      break;
    case 16:
      furthest_point_sampling_kernel<16>
          <<<b, n_threads, 0, stream>>>(b, n, m, dataset, temp, idxs);
      break;
    case 8:
      furthest_point_sampling_kernel<8>
          <<<b, n_threads, 0, stream>>>(b, n, m, dataset, temp, idxs);
      break;
    case 4:
      furthest_point_sampling_kernel<4>
          <<<b, n_threads, 0, stream>>>(b, n, m, dataset, temp, idxs);
      break;
    case 2:
      furthest_point_sampling_kernel<2>
          <<<b, n_threads, 0, stream>>>(b, n, m, dataset, temp, idxs);
      break;
    case 1:
      furthest_point_sampling_kernel<1>
          <<<b, n_threads, 0, stream>>>(b, n, m, dataset, temp, idxs);
      break;
    default:
      furthest_point_sampling_kernel<512>
          <<<b, n_threads, 0, stream>>>(b, n, m, dataset, temp, idxs);
  }
  CUDA_CHECK_ERRORS();
}



// input: points(b, c, n) idx(b, npoints, nsample)
// output: out(b, c, npoints, nsample)
__global__ void group_points_kernel(int b, int c, int n, int npoints,
                                    int nsample,
                                    const float *__restrict__ points,
                                    const int *__restrict__ idx,
                                    float *__restrict__ out) {
  int batch_index = blockIdx.x;
  points += batch_index * n * c;
  idx += batch_index * npoints * nsample;
  out += batch_index * npoints * nsample * c;

  const int index = threadIdx.y * blockDim.x + threadIdx.x;
  const int stride = blockDim.y * blockDim.x;
  for (int i = index; i < c * npoints; i += stride) {
    const int l = i / npoints;
    const int j = i % npoints;
    for (int k = 0; k < nsample; ++k) {
      int ii = idx[j * nsample + k];
      out[(l * npoints + j) * nsample + k] = points[l * n + ii];
    }
  }
}

void GroupPointsKernelLauncher(const int b, const int c, const int n, const int npoints, const int nsample,
                               const float *points, const int *idx, float *out) {
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  group_points_kernel<<<b, opt_block_config(npoints, c), 0, stream>>>(
      b, c, n, npoints, nsample, points, idx, out);

  CUDA_CHECK_ERRORS();
}

// input: grad_out(b, c, npoints, nsample), idx(b, npoints, nsample)
// output: grad_points(b, c, n)
__global__ void group_points_grad_kernel(int b, int c, int n, int npoints,
                                         int nsample,
                                         const float *__restrict__ grad_out,
                                         const int *__restrict__ idx,
                                         float *__restrict__ grad_points) {
  int batch_index = blockIdx.x;
  grad_out += batch_index * npoints * nsample * c;
  idx += batch_index * npoints * nsample;
  grad_points += batch_index * n * c;

  const int index = threadIdx.y * blockDim.x + threadIdx.x;
  const int stride = blockDim.y * blockDim.x;
  for (int i = index; i < c * npoints; i += stride) {
    const int l = i / npoints;
    const int j = i % npoints;
    for (int k = 0; k < nsample; ++k) {
      int ii = idx[j * nsample + k];
      atomicAdd(grad_points + l * n + ii,
                grad_out[(l * npoints + j) * nsample + k]);
    }
  }
}

void GroupPointsGradKernelLauncher(const int b, const int c, const int n, const int npoints,
                                   const int nsample, const float *grad_out, const int *idx, float *grad_points) {
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  group_points_grad_kernel<<<b, opt_block_config(npoints, c), 0, stream>>>(
      b, c, n, npoints, nsample, grad_out, idx, grad_points);

  CUDA_CHECK_ERRORS();
}



// input: unknown(b, n, 3) known(b, m, 3)
// output: dist2(b, n, 3), idx(b, n, 3)
__global__ void three_nn_kernel(int b, int n, int m,
                                const float *__restrict__ unknown,
                                const float *__restrict__ known,
                                float *__restrict__ dist2,
                                int *__restrict__ idx) {
  int batch_index = blockIdx.x;
  unknown += batch_index * n * 3;
  known += batch_index * m * 3;
  dist2 += batch_index * n * 3;
  idx += batch_index * n * 3;

  int index = threadIdx.x;
  int stride = blockDim.x;
  for (int j = index; j < n; j += stride) {
    float ux = unknown[j * 3 + 0];
    float uy = unknown[j * 3 + 1];
    float uz = unknown[j * 3 + 2];

    double best1 = 1e40, best2 = 1e40, best3 = 1e40;
    int besti1 = 0, besti2 = 0, besti3 = 0;
    for (int k = 0; k < m; ++k) {
      float x = known[k * 3 + 0];
      float y = known[k * 3 + 1];
      float z = known[k * 3 + 2];
      float d = (ux - x) * (ux - x) + (uy - y) * (uy - y) + (uz - z) * (uz - z);
      if (d < best1) {
        best3 = best2;
        besti3 = besti2;
        best2 = best1;
        besti2 = besti1;
        best1 = d;
        besti1 = k;
      } else if (d < best2) {
        best3 = best2;
        besti3 = besti2;
        best2 = d;
        besti2 = k;
      } else if (d < best3) {
        best3 = d;
        besti3 = k;
      }
    }
    dist2[j * 3 + 0] = best1;
    dist2[j * 3 + 1] = best2;
    dist2[j * 3 + 2] = best3;

    idx[j * 3 + 0] = besti1;
    idx[j * 3 + 1] = besti2;
    idx[j * 3 + 2] = besti3;
  }
}

void ThreeNNKernelLauncher(int b, int n, int m, const float *unknown,
                             const float *known, float *dist2, int *idx) {
  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  three_nn_kernel<<<b, opt_n_threads(n), 0, stream>>>(b, n, m, unknown, known,
                                                      dist2, idx);

  CUDA_CHECK_ERRORS();
}

// input: points(b, c, m), idx(b, n, 3), weight(b, n, 3)
// output: out(b, c, n)
__global__ void three_interpolate_kernel(int b, int c, int m, int n,
                                         const float *__restrict__ points,
                                         const int *__restrict__ idx,
                                         const float *__restrict__ weight,
                                         float *__restrict__ out) {
  int batch_index = blockIdx.x;
  points += batch_index * m * c;

  idx += batch_index * n * 3;
  weight += batch_index * n * 3;

  out += batch_index * n * c;

  const int index = threadIdx.y * blockDim.x + threadIdx.x;
  const int stride = blockDim.y * blockDim.x;
  for (int i = index; i < c * n; i += stride) {
    const int l = i / n;
    const int j = i % n;
    float w1 = weight[j * 3 + 0];
    float w2 = weight[j * 3 + 1];
    float w3 = weight[j * 3 + 2];

    int i1 = idx[j * 3 + 0];
    int i2 = idx[j * 3 + 1];
    int i3 = idx[j * 3 + 2];

    out[i] = points[l * m + i1] * w1 + points[l * m + i2] * w2 +
             points[l * m + i3] * w3;
  }
}

void ThreeInterpolateKernelLauncher(int b, int c, int m, int n,
                                      const float *points, const int *idx,
                                      const float *weight, float *out) {
  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  three_interpolate_kernel<<<b, opt_block_config(n, c), 0, stream>>>(
      b, c, m, n, points, idx, weight, out);

  CUDA_CHECK_ERRORS();
}

// input: grad_out(b, c, n), idx(b, n, 3), weight(b, n, 3)
// output: grad_points(b, c, m)

__global__ void three_interpolate_grad_kernel(
    int b, int c, int n, int m, const float *__restrict__ grad_out,
    const int *__restrict__ idx, const float *__restrict__ weight,
    float *__restrict__ grad_points) {
  int batch_index = blockIdx.x;
  grad_out += batch_index * n * c;
  idx += batch_index * n * 3;
  weight += batch_index * n * 3;
  grad_points += batch_index * m * c;

  const int index = threadIdx.y * blockDim.x + threadIdx.x;
  const int stride = blockDim.y * blockDim.x;
  for (int i = index; i < c * n; i += stride) {
    const int l = i / n;
    const int j = i % n;
    float w1 = weight[j * 3 + 0];
    float w2 = weight[j * 3 + 1];
    float w3 = weight[j * 3 + 2];

    int i1 = idx[j * 3 + 0];
    int i2 = idx[j * 3 + 1];
    int i3 = idx[j * 3 + 2];

    atomicAdd(grad_points + l * m + i1, grad_out[i] * w1);
    atomicAdd(grad_points + l * m + i2, grad_out[i] * w2);
    atomicAdd(grad_points + l * m + i3, grad_out[i] * w3);
  }
}

void ThreeInterpolateGradKernelLauncher(int b, int c, int n, int m,
                                           const float *grad_out,
                                           const int *idx, const float *weight,
                                           float *grad_points) {
  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  three_interpolate_grad_kernel<<<b, opt_block_config(n, c), 0, stream>>>(
      b, c, n, m, grad_out, idx, weight, grad_points);

  CUDA_CHECK_ERRORS();
}



// input: new_xyz(b, m, 3) xyz(b, n, 3)
// output: idx(b, m, nsample)
__global__ void query_ball_point_kernel(const int b, const int n, const int m, const float radius, const int nsample,
                                             const float *__restrict__ new_xyz, const float *__restrict__ xyz,
                                             int *__restrict__ idx) {
  int batch_index = blockIdx.x;
  xyz += batch_index * n * 3;
  new_xyz += batch_index * m * 3;
  idx += m * nsample * batch_index;

  int index = threadIdx.x;
  int stride = blockDim.x;

  float radius2 = radius * radius;
  for (int j = index; j < m; j += stride) {
    float new_x = new_xyz[j * 3 + 0];
    float new_y = new_xyz[j * 3 + 1];
    float new_z = new_xyz[j * 3 + 2];
    for (int k = 0, cnt = 0; k < n && cnt < nsample; ++k) {
      float x = xyz[k * 3 + 0];
      float y = xyz[k * 3 + 1];
      float z = xyz[k * 3 + 2];
      float d2 = (new_x - x) * (new_x - x) + (new_y - y) * (new_y - y) +
                 (new_z - z) * (new_z - z);
      if (d2 < radius2) {
        if (cnt == 0) {
          for (int l = 0; l < nsample; ++l) {
            idx[j * nsample + l] = k;
          }
        }
        idx[j * nsample + cnt] = k;
        ++cnt;
      }
    }
  }
}

void QueryBallPointKernelLauncher(const int b, const int n, const int m, const float radius, const int nsample, const float *new_xyz, const float *xyz, int *idx) {
    hipStream_t stream = at::cuda::getCurrentCUDAStream();
    query_ball_point_kernel<<<b, opt_n_threads(m), 0, stream>>>(b, n, m, radius, nsample, new_xyz, xyz, idx);
    CUDA_CHECK_ERRORS();
}
